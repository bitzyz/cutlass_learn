#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdarg.h>
#include <stdio.h>
#include <hipblas.h>
#include <stdlib.h>
#include <cute/tensor.hpp>
#include <float.h>
#include "utils.h"

using T = cute::half_t;
using namespace cute;

template <typename T>
__global__ void gpu_compare_kernel(const T *x, const T *y, int n,
                                   float threshold, int *count,
                                   float *max_error)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= n)
    {
        return;
    }

    float v0 = x[idx];
    float v1 = y[idx];

    float diff = fabs(v0 - v1);
    if (diff > threshold)
    {
        atomicAdd(count, 1);

        // for positive floating point, there int representation is in the same
        // order.
        int int_diff = *((int *)(&diff));
        atomicMax((int *)max_error, int_diff);
    }
}

template <typename T>
void compare(const T *x, const T *y, int n, float threshold)
{

    int *num_count, *num_count_h;
    float *max_error, *max_error_h;
    num_count_h = (int *)malloc(sizeof(int));
    max_error_h = (float *)malloc(sizeof(float));
    hipMalloc(&num_count, sizeof(int));
    hipMalloc(&max_error, sizeof(float));
    hipMemset(num_count, 0, sizeof(int));
    hipMemset(max_error, 0.0, sizeof(float));

    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);
    gpu_compare_kernel<<<grid, block>>>(x, y, n, threshold, num_count, max_error);
    hipDeviceSynchronize();
    hipMemcpy(num_count_h, num_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(max_error_h, max_error, sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    int num = *num_count_h;
    float error = *max_error_h;

    if (num == 0)
    {
        printf("check ok, max_error = %f\n", error);
    }
    else
    {
        float p = (100.f * num) / n;
        printf("===============================\n");
        printf("check fail: diff %.1f%% = %d/%d max_error = %f\n", p, num, n,
               error);
        printf("===============================\n");
    }
}

template <typename T, int BM, int BN, int BK, int kStage, typename TiledMMA,
          typename G2SCopyA, typename G2SCopyB,
          typename SmemLayoutA, typename SmemLayoutB, typename SmemLayoutC,
          typename S2RCopyAtomA, typename S2RCopyAtomB,
          typename R2SCopyAtomC, typename S2GCopyAtomC, typename S2GCopyC>
__global__ void gemm_epilogue_v4(const T *Aptr, const T *Bptr, T *Dptr, int m, int n, int k)
{
    // Initilize shared memory
    extern __shared__ T shm_data[];

    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

    // Initilize thread block
    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    // use Tensor notation to represent device pointer + dimension
    Tensor A = make_tensor(make_gmem_ptr(Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr(Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor D = make_tensor(make_gmem_ptr(Dptr), make_shape(m, n), make_stride(n, Int<1>{}));

    // slice the tensor to small one which is used for current thread block.
    Tensor gA = local_tile(A, make_tile(Int<BM>{}, Int<BK>{}), make_coord(iy, _));  // (BM, BK, num_tile_k)
    Tensor gB = local_tile(B, make_tile(Int<BN>{}, Int<BK>{}), make_coord(ix, _));  // (BN, BK, num_tile_k)
    Tensor gD = local_tile(D, make_tile(Int<BM>{}, Int<BN>{}), make_coord(iy, ix)); // (BM, BN)

    // shared memory
    auto sA = make_tensor(make_smem_ptr(Ashm),
                          SmemLayoutA{});                      // (BM, BK, kStage)
    auto sB = make_tensor(make_smem_ptr(Bshm), SmemLayoutB{}); // (BN, BK, kStage)

    // dispatch TileA/TileB/TileC mma tensor into thread fragment via partition
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(threadIdx.x);
    // auto tCsA = thr_mma.partition_A(sA);                               // (MMA,MMA_M,MMA_K,kStage)
    // auto tCsB = thr_mma.partition_B(sB);                               // (MMA,MMA_N,MMA_K,kStage)
    auto tCgD = thr_mma.partition_C(gD); // (MMA, MMA_M, MMA_N)

    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0)); // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0)); // (MMA, MMA_N, MMA_K)
    auto tCrD = thr_mma.partition_fragment_C(gD);          // (MMA, MMA_M, MMA_N)
    clear(tCrD);

    // from global memory to shared memory
    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA); // (CPY, CPY_M, CPY_K, k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA); // (CPY, CPY_M, CPY_K, kStage)

    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB); // (CPY, CPY_N, CPY_K, k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB); // (CPY, CPY_N, CPY_K, kStage)

    // from shared memory to register, use tiled_mma to generate tiled_copy
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tAsA = s2r_thr_copy_a.partition_S(sA);     // (CPY, CPY_M, CPY_K, kStage)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA); // (CPY, CPY_M, CPY_K)

    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    auto tBsB = s2r_thr_copy_b.partition_S(sB);     // (CPY, CPY_N, CPY_K, kStage)
    auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB); // (CPY, CPY_N, CPY_K)

    /* PREFETCH */
    // submit kStage - 1 tile
    // gmem -> shm
    int itile_to_read = 0;
    int ismem_read = 0;
    int ismem_write = 0;

#pragma unroll
    for (int istage = 0; istage < kStage - 1; ++istage)
    {
        cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, istage),
                   tAsA_copy(_, _, _, istage));
        cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, istage),
                   tBsB_copy(_, _, _, istage));
        cp_async_fence();

        ++itile_to_read;
        ++ismem_write;
    }

    // wait one submitted gmem->smem done
    cp_async_wait<kStage - 2>();
    __syncthreads();

    int ik = 0;
    // smem -> reg
    cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik, ismem_read), tCrA_view(_, _, ik));
    cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik, ismem_read), tCrB_view(_, _, ik));

    // loop over k: i. load tile, ii. mma
    int ntile = k / BK;
#pragma unroll 1
    for (int itile = 0; itile < ntile; ++itile)
    {
        int nk = size<2>(tCrA);

#pragma unroll
        for (int ik = 0; ik < nk; ++ik)
        {
            int ik_next = (ik + 1) % nk;

            if (ik == nk - 1)
            {
                cp_async_wait<kStage - 2>();
                __syncthreads();

                ismem_read = (ismem_read + 1) % kStage;
            }

            // shm -> reg s[itile][ik + 1] -> r[ik + 1]
            cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik_next, ismem_read),
                       tCrA_view(_, _, ik_next));
            cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik_next, ismem_read),
                       tCrB_view(_, _, ik_next));

            if (ik == 0)
            {
                if (itile_to_read < ntile)
                {
                    cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile_to_read),
                               tAsA_copy(_, _, _, ismem_write));
                    cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile_to_read),
                               tBsB_copy(_, _, _, ismem_write));

                    ++itile_to_read;
                    ismem_write = (ismem_write + 1) % kStage;
                }

                cp_async_fence();
            }

            cute::gemm(tiled_mma, tCrD, tCrA(_, _, ik), tCrB(_, _, ik), tCrD);
        } // for ik
    }

    // use less shared memory as a scratchpad tile to use large wide instuction
    // Dreg -> shm -> reg -> global
    auto sC = make_tensor(sA(_, _, ismem_read).data(), SmemLayoutC{});

    auto r2s_tiled_copy_c = make_tiled_copy_C(R2SCopyAtomC{}, tiled_mma);
    auto r2s_thr_copy_c = r2s_tiled_copy_c.get_slice(idx);
    auto tCrC_r2s = r2s_thr_copy_c.retile_S(tCrD);  // (CPY, CPY_M, CPY_N)
    auto tCsC_r2s = r2s_thr_copy_c.partition_D(sC); // (CPY, _1, _1, pipe)

    S2GCopyC s2g_tiled_copy_c;
    auto s2g_thr_copy_c = s2g_tiled_copy_c.get_thread_slice(idx);
    auto tCsC_s2g = s2g_thr_copy_c.partition_S(sC); // (CPY, _1, _1, pipe)
    auto tCgC_s2g = s2g_thr_copy_c.partition_D(gD); // (CPY, CPY_M, CPY_N)

    auto tCgC_s2gx = group_modes<1, 3>(tCgC_s2g); // (CPY_, CPY_MN)
    auto tCrC_r2sx = group_modes<1, 3>(tCrC_r2s); // (CPY_, CPY_MN)

    int step = size<3>(tCsC_r2s); // pipe
#pragma unroll
    for (int i = 0; i < size<1>(tCrC_r2sx); i += step)
    {
        // reg -> shm
#pragma unroll
        for (int j = 0; j < step; ++j)
        {
            // we add a temp tensor to cope with accumulator and output data type
            // difference
            auto t = make_tensor_like<T>(tCrC_r2sx(_, i + j));
            cute::copy(tCrC_r2sx(_, i + j), t);

            cute::copy(r2s_tiled_copy_c, t, tCsC_r2s(_, 0, 0, j));
        }
        __syncthreads();

#pragma unroll
        // shm -> global
        for (int j = 0; j < step; ++j)
        {
            cute::copy(s2g_tiled_copy_c, tCsC_s2g(_, 0, 0, j), tCgC_s2gx(_, i + j));
        }

        __syncthreads();
    }
}

template <typename T>
void gemm_v4(T *a, T *b, T *c, int M, int N, int K)
{
    auto BM = Int<32>{};
    auto BN = Int<256>{};
    auto BK = Int<256>{};
    auto KStage = Int<4>{};
    auto kSmemLayoutCBatch = Int<2>{};

    // Define the smem layouts
    using SmemLayoutAtom = decltype(composition(
        Swizzle<3, 3, 3>{},
        make_layout(make_shape(Int<8>{}, Int<BK>{}),
                    make_stride(Int<BK>{}, Int<1>{}))));
    using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BM>{}, Int<BK>{}, Int<KStage>{})));
    using SmemLayoutB = decltype(tile_to_shape(SmemLayoutAtom{},
                                               make_shape(Int<BN>{}, Int<BK>{}, Int<KStage>{}))); // (m,n) -> smem_idx

    // mma
    using mma_op = SM80_16x8x16_F16F16F16F16_TN;
    using mma_traits = MMA_Traits<mma_op>;
    using mma_atom = MMA_Atom<mma_traits>;
    static constexpr int kMmaEURepeatM = 1;
    static constexpr int kMmaEURepeatN = 2;
    static constexpr int kMmaEURepeatK = 2;

    using mma_atom_shape = mma_traits::Shape_MNK;
    static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
    static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
    static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});
    using MMA_EU_RepeatT = decltype(make_layout(make_shape(
        Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
    using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;
    using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));

    // copy from global memory to shared memory
    using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
    using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
    using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;
    using G2SCopyA =
        decltype(make_tiled_copy(g2s_copy_atom{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}), // Thr layout 32x4 k-major
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{})))); // Val layout 1x8
    using G2SCopyB = G2SCopyA;

    // copy from shared memory to register
    // use mma tiled ,so no tiled here
    using s2r_copy_op = SM75_U32x4_LDSM_N;
    using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
    using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;
    using S2RCopyAtomA = s2r_copy_atom;
    using S2RCopyAtomB = s2r_copy_atom;

    // epilogue: register to global via shared memory
    using SmemLayoutAtomC = decltype(composition(
        Swizzle<3, 3, 3>{}, make_layout(make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}),
                                        make_stride(Int<kMmaPN>{}, Int<1>{}))));
    using SmemLayoutC = decltype(tile_to_shape(
        SmemLayoutAtomC{},
        make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}, Int<kSmemLayoutCBatch>{})));

    static_assert(size<0>(SmemLayoutA{}) * size<1>(SmemLayoutA{}) >=
                      size(SmemLayoutC{}),
                  "C shared memory request is large than A's one pipe");

    using R2SCopyAtomC = Copy_Atom<UniversalCopy<int>, T>;

    using S2GCopyAtomC = Copy_Atom<UniversalCopy<cute::uint128_t>, T>;
    using S2GCopyC =
        decltype(make_tiled_copy(S2GCopyAtomC{},
                                 make_layout(make_shape(Int<32>{}, Int<4>{}),
                                             make_stride(Int<4>{}, Int<1>{})),
                                 make_layout(make_shape(Int<1>{}, Int<8>{}))));

    int BX = (N + BN - 1) / BN;
    int BY = (M + BM - 1) / BM;

    dim3 block(size(MMA{}));
    dim3 grid(BX, BY);

    // C_shm is shared with A_shm and B_shm
    static constexpr int shm_size_AB =
        cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
    static constexpr int shm_size_C = cute::cosize(SmemLayoutC{});
    static constexpr int kShmSize =
        cute::max(shm_size_AB, shm_size_C) * sizeof(T);

    int shm_size = kShmSize;

    hipFuncSetAttribute(reinterpret_cast<const void*>(gemm_epilogue_v4<T), BM, BN, BK, KStage, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, SmemLayoutC, S2RCopyAtomA, S2RCopyAtomB, R2SCopyAtomC, S2GCopyAtomC, S2GCopyC>,
                         hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);

    gemm_epilogue_v4<T, BM, BN, BK, KStage, MMA, G2SCopyA, G2SCopyB, SmemLayoutA, SmemLayoutB, SmemLayoutC, S2RCopyAtomA, S2RCopyAtomB, R2SCopyAtomC, S2GCopyAtomC, S2GCopyC>
        <<<grid, block, shm_size>>>(a, b, c, M, N, K);
}

int main()
{
    const int repeat = 100;

    printf("\nalgo = Cute_HGEMM_V4\n");

    // const int M = 256, N = 256, K = 256;
    const int M = 32, N = 32768, K = 8192;
    testF16F16GemmMaxError<T>(
        gemm_v4, compare, M, N, K, repeat);

    // testF16F16GemmPerformance<T>(
    //     gemm_v1, M, N, K, repeat);

    return 0;
}