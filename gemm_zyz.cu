#include "hip/hip_runtime.h"
#include <cstdarg>
#include <cute/tensor.hpp>
#include <iostream>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hipblas.h>

template <typename T>
void gpu_compare(const T *x, const T *y, int n, float threshold = 1.E-1);

template <typename T>
void cpu_rand_data(T *c);

template <typename Config>
__global__ void
gemm_multi_stage(void *Dptr, const void *Aptr, const void *Bptr, int m, int n, int k)
{
    using namespace cute;
    // using X = Underscore;
    using T = typename Config::T;
    using SmemLayoutA = typename Config::SmemLayoutA;
    using SmemLayoutB = typename Config::SmemLayoutB;
    using SmemLayoutC = typename Config::SmemLayoutC;
    using TiledMMA = typename Config::MMA;

    using S2RCopyAtomA = typename Config::S2RCopyAtomA;
    using S2RCopyAtomB = typename Config::S2RCopyAtomB;
    using G2SCopyA = typename Config::G2SCopyA;
    using G2SCopyB = typename Config::G2SCopyB;
    using R2SCopyAtomC = typename Config::R2SCopyAtomC;
    using S2GCopyAtomC = typename Config::S2GCopyAtomC;
    using S2GCopyC = typename Config::S2GCopyC;

    constexpr int kTileM = Config::kTileM;
    constexpr int kTileN = Config::kTileN;
    constexpr int kTileK = Config::kTileK;
    constexpr int kStage = Config::kStage;

    extern __shared__ T shm_data[];

    T *Ashm = shm_data;
    T *Bshm = shm_data + cute::cosize(SmemLayoutA{});

    int idx = threadIdx.x;
    int ix = blockIdx.x;
    int iy = blockIdx.y;

    // make tensor
    Tensor A = make_tensor(make_gmem_ptr((T *)Aptr), make_shape(m, k), make_stride(k, Int<1>{}));
    Tensor B = make_tensor(make_gmem_ptr((T *)Bptr), make_shape(n, k), make_stride(k, Int<1>{}));
    Tensor D = make_tensor(make_gmem_ptr((T *)Dptr), make_shape(m, n), make_stride(n, Int<1>{}));

    // slice tensor to small one which is used for current thread block
    Tensor gA = local_tile(A, make_tile(Int<kTileM>{}, Int<kTileK>{}), make_coord(iy, _));  // (kTileM, kTileK, k)
    Tensor gB = local_tile(B, make_tile(Int<kTileN>{}, Int<kTileK>{}), make_coord(ix, _));  // (kTileN, kTileK, k)
    Tensor gD = local_tile(D, make_tile(Int<kTileM>{}, Int<kTileN>{}), make_coord(iy, ix)); // (kTileM, kTileN)

    // shared memory
    auto sA = make_tensor(make_smem_ptr(Ashm), SmemLayoutA{}); // (kTileM, kTileK, kStage)
    auto sB = make_tensor(make_smem_ptr(Bshm), SmemLayoutB{}); // (kTileN, kTileK, kStage)

    // dispatch TileA/TileB/TileD mma tensor into thread fragment via partition
    TiledMMA tiled_mma;
    auto thr_mma = tiled_mma.get_slice(idx);
    auto tCrA = thr_mma.partition_fragment_A(gA(_, _, 0)); // (MMA, MMA_M, MMA_K)
    auto tCrB = thr_mma.partition_fragment_B(gB(_, _, 0)); // (MMA, MMA_N, MMA_K)
    auto tCrD = thr_mma.partition_fragment_C(gD);          // (MMA, MMA_M, MMA_N)

    // fill zero for accumulator
    clear(tCrD);

    // gemm -cp.async -> shm -ldmatrix -> reg
    auto s2r_tiled_copy_a = make_tiled_copy_A(S2RCopyAtomA{}, tiled_mma);
    auto s2r_thr_copy_a = s2r_tiled_copy_a.get_slice(idx);
    auto tAsA = s2r_thr_copy_a.partition_S(sA);     // (CPY, CPY_M, CPY_K, kStage)
    auto tCrA_view = s2r_thr_copy_a.retile_D(tCrA); // (CPY, CPY_M, CPY_K)

    auto s2r_tiled_copy_b = make_tiled_copy_B(S2RCopyAtomB{}, tiled_mma);
    auto s2r_thr_copy_b = s2r_tiled_copy_b.get_slice(idx);
    auto tBsB = s2r_thr_copy_b.partition_S(sB);     // (CPY, CPY_N, CPY_K, kStage)
    auto tCrB_view = s2r_thr_copy_b.retile_D(tCrB); // (CPY, CPY_N, CPY_K)

    G2SCopyA g2s_tiled_copy_a;
    auto g2s_thr_copy_a = g2s_tiled_copy_a.get_slice(idx);
    auto tAgA_copy = g2s_thr_copy_a.partition_S(gA); // (CPY, CPY_M, CPY_K, k)
    auto tAsA_copy = g2s_thr_copy_a.partition_D(sA); // (CPY, CPY_M, CPY_K, kStage)

    G2SCopyB g2s_tiled_copy_b;
    auto g2s_thr_copy_b = g2s_tiled_copy_b.get_slice(idx);
    auto tBgB_copy = g2s_thr_copy_b.partition_S(gB); // (CPY, CPY_N, CPY_K, k)
    auto tBsB_copy = g2s_thr_copy_b.partition_D(sB); // (CPY, CPY_N, CPY_K, kStage)

    int itile_to_read = 0;
    int ismem_read = 0;
    int ismem_write = 0;

// gmem -> shm
#pragma unroll
    for (int istage = 0; istage < kStage - 1; ++istage)
    {
        cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, istage), tAsA_copy(_, _, _, istage));
        cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, istage), tBsB_copy(_, _, _, istage));
        cp_async_fence();
        ++itile_to_read;
        ++ismem_write;
    }

    // wait one submitted gmem -> shm done
    cp_async_wait<kStage - 2>();
    __syncthreads();

    int ik = 0;
    // shm -> reg
    cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik, ismem_read), tCrA_view(_, _, ik));
    cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik, ismem_read), tCrB_view(_, _, ik));

    // loop over k (1. load tile 2. mma)
    int ntile = k / kTileK;
#pragma unroll 1
    for (int itile = 0; itile < ntile; ++itile)
    {
        int nk = size<2>(tCrA);

#pragma unroll
        for (int ik = 0; ik < nk; ++ik)
        {
            int ik_next = (ik + 1) % nk;
            if (ik == nk - 1)
            {
                cp_async_wait<kStage - 2>();
                __syncthreads();

                ismem_read = (ismem_read + 1) % kStage;
            }

            // shm -> reg s[itile][ik + 1] -> r[ik+1]
            cute::copy(s2r_tiled_copy_a, tAsA(_, _, ik_next, ismem_read), tCrA_view(_, _, ik_next));
            cute::copy(s2r_tiled_copy_b, tBsB(_, _, ik_next, ismem_read), tCrB_view(_, _, ik_next));
            if (ik == 0)
            {
                if (itile_to_read < ntile)
                {
                    cute::copy(g2s_tiled_copy_a, tAgA_copy(_, _, _, itile_to_read),
                               tAsA_copy(_, _, _, ismem_write));
                    cute::copy(g2s_tiled_copy_b, tBgB_copy(_, _, _, itile_to_read),
                               tBsB_copy(_, _, _, ismem_write));

                    ++itile_to_read;
                    ismem_write = (ismem_write + 1) % kStage;
                }

                cp_async_fence();
            }
            // 2. mma
            cute::gemm(tiled_mma, tCrD, tCrA(_, _, ik), tCrB(_, _, ik), tCrD);
        }
    }
    // use less shared memory as a scratchpad tile to use large wide instuction
    // Dreg -> shm -> reg -> global
    auto sC = make_tensor(sA(_, _, ismem_read).data(), SmemLayoutC{});
    auto r2s_tiled_copy_c = make_tiled_copy_C(R2SCopyAtomC{}, tiled_mma);
    auto r2s_thr_copy_c = r2s_tiled_copy_c.get_slice(idx);
    auto tCrC_r2s = r2s_thr_copy_c.retile_S(tCrD);  //(CPY, CPY_M, CPY_N)
    auto tCsC_r2s = r2s_thr_copy_c.partition_D(sC); // (CPY, _1, _1, pipe)

    S2GCopyC s2g_tiled_copy_c;
    auto s2g_thr_copy_c = s2g_tiled_copy_c.get_thread_slice(idx);
    auto tCsC_s2g = s2g_thr_copy_c.partition_S(sC); // (CPY, _1, _1, pipe)
    auto tCgC_s2g = s2g_thr_copy_c.partition_D(gD); // (CPY, CPY_M, CPY_N)

    auto tCgC_s2gx = group_modes<1, 3>(tCgC_s2g); // (CPY_, CPY_MN)
    auto tCrC_r2sx = group_modes<1, 3>(tCrC_r2s); // (CPY_, CPY_MN)

    int step = size<3>(tCsC_r2s);
#pragma unroll
    for (int i = 0; i < size<1>(tCrC_r2sx); i += step)
    {
        // reg -> shm
#pragma unroll
        for (int j = 0; j < step; ++j)
        {
            // we add a temp tensor to cope with accumulator and output data type
            // difference
            auto t = make_tensor_like<T>(tCrC_r2sx(_, i + j));
            cute::copy(tCrC_r2sx(_, i + j), t);

            cute::copy(r2s_tiled_copy_c, t, tCsC_r2s(_, 0, 0, j));
        }
        __syncthreads();

#pragma unroll
        // shm -> global
        for (int j = 0; j < step; ++j)
        {
            cute::copy(s2g_tiled_copy_c, tCsC_s2g(_, 0, 0, j), tCgC_s2gx(_, i + j));
        }

        __syncthreads();
    }
}

namespace config
{
    using namespace cute;
    template <typename T_, int kTileM_ = 128, int kTileN_ = 128, int kTileK_ = 32,
              int kStage_ = 5, int kSmemLayoutCBatch_ = 2,
              typename ComputeType = T_>
    struct GemmConfig
    {
        using T = T_;
        // tile config
        static constexpr int kTileM = kTileM_;
        static constexpr int kTileN = kTileN_;
        static constexpr int kTileK = kTileK_;
        static constexpr int kStage = kStage_;
        static constexpr int kSmemLayoutCBatch = kSmemLayoutCBatch_;

        static constexpr int kShmLoadSwizzleM = 3;
        static constexpr int kShmLoadSwizzleS = 3;
        static constexpr int kShmLoadSwizzleB = 3;

        using SmemLayoutAtom = decltype(composition(Swizzle<kShmLoadSwizzleB, kShmLoadSwizzleM, kShmLoadSwizzleS>{},
                                                    make_layout(make_shape(Int<8>{}, Int<kTileK>{}),
                                                                make_stride(Int<kTileK>{}, Int<1>{}))));
        using SmemLayoutA = decltype(tile_to_shape(SmemLayoutAtom{},
                                                   make_shape(Int<kTileM>{}, Int<kTileK>{}, Int<kStage>{})));
        using SmemLayoutB = decltype(tile_to_shape(SmemLayoutAtom{},
                                                   make_shape(Int<kTileN>{}, Int<kTileK>{}, Int<kStage>{})));

        using mma_op = SM80_16x8x16_F16F16F16F16_TN;
        using mma_traits = MMA_Traits<mma_op>;
        using mma_atom = MMA_Atom<mma_traits>;

        static constexpr int kMmaEURepeatM = 2; // 2
        static constexpr int kMmaEURepeatN = 2;
        static constexpr int kMmaEURepeatK = 1;

        using mma_atom_shape = mma_traits::Shape_MNK;
        static constexpr int kMmaPM = 1 * kMmaEURepeatM * get<0>(mma_atom_shape{});
        static constexpr int kMmaPN = 2 * kMmaEURepeatN * get<1>(mma_atom_shape{});
        static constexpr int kMmaPK = 1 * kMmaEURepeatK * get<2>(mma_atom_shape{});

        using MMA_EU_RepeatT = decltype(make_layout(make_shape(
            Int<kMmaEURepeatM>{}, Int<kMmaEURepeatN>{}, Int<kMmaEURepeatK>{})));
        using MMA_P_T = Tile<Int<kMmaPM>, Int<kMmaPN>, Int<kMmaPK>>;

        using MMA = decltype(make_tiled_mma(mma_atom{}, MMA_EU_RepeatT{}, MMA_P_T{}));

        // global to shared memory copy
        using g2s_copy_op = SM80_CP_ASYNC_CACHEGLOBAL<cute::uint128_t>;
        using g2s_copy_traits = Copy_Traits<g2s_copy_op>;
        using g2s_copy_atom = Copy_Atom<g2s_copy_traits, T>;

        using G2SCopyA =
            decltype(make_tiled_copy(g2s_copy_atom{},
                                     make_layout(make_shape(Int<32>{}, Int<4>{}),
                                                 make_stride(Int<4>{}, Int<1>{})),
                                     make_layout(make_shape(Int<1>{}, Int<8>{}))));
        using G2SCopyB = G2SCopyA;

        // shared memory to register copy
        using s2r_copy_op = SM75_U32x4_LDSM_N;
        using s2r_copy_traits = Copy_Traits<s2r_copy_op>;
        using s2r_copy_atom = Copy_Atom<s2r_copy_traits, T>;

        using S2RCopyAtomA = s2r_copy_atom;
        using S2RCopyAtomB = s2r_copy_atom;

        // epilogue: register to global via shared memory
        using SmemLayoutAtomC = decltype(composition(
            Swizzle<2, 3, 3>{}, make_layout(make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}),
                                            make_stride(Int<kMmaPN>{}, Int<1>{}))));
        using SmemLayoutC = decltype(tile_to_shape(
            SmemLayoutAtomC{},
            make_shape(Int<kMmaPM>{}, Int<kMmaPN>{}, Int<kSmemLayoutCBatch>{})));

        static_assert(size<0>(SmemLayoutA{}) * size<1>(SmemLayoutA{}) >=
                          size(SmemLayoutC{}),
                      "C shared memory request is large than A's one pipe");

        using R2SCopyAtomC = Copy_Atom<UniversalCopy<int>, T>;

        using S2GCopyAtomC = Copy_Atom<UniversalCopy<cute::uint128_t>, T>;
        using S2GCopyC =
            decltype(make_tiled_copy(S2GCopyAtomC{},
                                     make_layout(make_shape(Int<32>{}, Int<4>{}),
                                                 make_stride(Int<4>{}, Int<1>{})),
                                     make_layout(make_shape(Int<1>{}, Int<8>{}))));

        static constexpr int kThreadNum = size(MMA{});
        static constexpr int shm_size_AB =
            cute::cosize(SmemLayoutA{}) + cute::cosize(SmemLayoutB{});
        static constexpr int shm_size_C = cute::cosize(SmemLayoutC{});

        static constexpr int kShmSize =
            cute::max(shm_size_AB, shm_size_C) * sizeof(T);
    };
} // namespace config

template <typename T>
void cpu_rand_data(T *c)
{
    auto t = *c;

    using ValueType = typename T::value_type;

    int n = size(t);
    for (int i = 0; i < n; ++i)
    {
        float v = ((rand() % 200) - 100.f) * 0.01f;
        // printf("v = %f\n", v);
        t(i) = ValueType(v);
    }
}
void printf_fail(const char *fmt, ...)
{
    int red = 31;
    int def = 39;

    printf("\033[%dm", red);

    va_list args;
    va_start(args, fmt);
    vprintf(fmt, args);
    va_end(args);

    printf("\033[%dm", def);
}

void printf_ok(const char *fmt, ...)
{
    int red = 32;
    int def = 39;

    printf("\033[%dm", red);

    va_list args;
    va_start(args, fmt);
    vprintf(fmt, args);
    va_end(args);

    printf("\033[%dm", def);
}

template <typename T>
__global__ void gpu_compare_kernel(const T *x, const T *y, int n,
                                   float threshold, int *count,
                                   float *max_error)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx >= n)
    {
        return;
    }

    float v0 = x[idx];
    float v1 = y[idx];

    float diff = fabs(v0 - v1);
    if (diff > threshold)
    {
        atomicAdd(count, 1);

        // for positive floating point, there int representation is in the same
        // order.
        int int_diff = *((int *)(&diff));
        atomicMax((int *)max_error, int_diff);
    }
}

template <typename T>
void gpu_compare(const T *x, const T *y, int n, float threshold)
{
    int *num_count;
    float *max_error;
    hipMalloc(&num_count, sizeof(int));
    hipMalloc(&max_error, sizeof(float));
    hipMemset(num_count, 0, sizeof(int));
    hipMemset(max_error, 0, sizeof(float));

    dim3 block(256);
    dim3 grid((n + block.x - 1) / block.x);
    gpu_compare_kernel<<<grid, block>>>(x, y, n, threshold, num_count, max_error);
    int num = 0;
    float error = 0;
    hipMemcpy(&num, num_count, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&error, max_error, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    if (num == 0)
    {
        printf_ok("check ok, max_error = %f\n", error);
    }
    else
    {
        float p = (100.f * num) / n;
        printf_fail("===============================\n");
        printf_fail("check fail: diff %.1f%% = %d/%d max_error = %f\n", p, num, n,
                    error);
        printf_fail("===============================\n");
    }
}

int main(int argc, char *argv[])
{
    using T = cute::half_t;
    using namespace cute;
    // using X = Underscore;

    srand(10086);

    int M = 81920; // 32
    int N = 256;   // 10240
    int K = 256;   // 8192
    std::cout << "##########################" << std::endl
              << "M = " << M << ", N = " << N << ", K = " << K << std::endl;

    using ComputeType = T;
    T *Aptr;
    T *Bptr;
    T *Dptr;
    T *Aptr_host;
    T *Bptr_host;
    T *Dptr_host;

    Aptr_host = (T *)malloc(sizeof(T) * M * K);
    Bptr_host = (T *)malloc(sizeof(T) * N * K);
    Dptr_host = (T *)malloc(sizeof(T) * M * N);

    hipMalloc(&Aptr, sizeof(T) * M * K);
    hipMalloc(&Bptr, sizeof(T) * N * K);
    hipMalloc(&Dptr, sizeof(T) * M * N);

    auto tA = make_tensor(Aptr_host, make_shape(M, K), make_stride(K, 1));
    auto tB = make_tensor(Bptr_host, make_shape(N, K), make_stride(K, 1));
    auto tD = make_tensor(Dptr_host, make_shape(M, N), make_stride(N, 1));
    std::cout << "###########################" << std::endl
              << "Run Cutlass Gemm:" << std::endl;

    cpu_rand_data(&tA);
    cpu_rand_data(&tB);
    clear(tD);

    hipMemcpy(Aptr, Aptr_host, sizeof(T) * M * K, hipMemcpyHostToDevice);
    hipMemcpy(Bptr, Bptr_host, sizeof(T) * N * K, hipMemcpyHostToDevice);
    hipMemcpy(Dptr, Dptr_host, sizeof(T) * M * N, hipMemcpyHostToDevice);

    config::GemmConfig<T, 128, 128, 32, 3> gemm_config;
    print(typename decltype(gemm_config)::MMA{});

    dim3 block = gemm_config.kThreadNum;
    dim3 grid((N + gemm_config.kTileN - 1) / gemm_config.kTileN,
              (M + gemm_config.kTileM - 1) / gemm_config.kTileM);
    int shm_size = gemm_config.kShmSize;

    // multi-stage
    hipMemset(Dptr, 0, sizeof(T) * M * N);
    hipFuncSetAttribute(gemm_multi_stage<decltype(gemm_config)>,
                         hipFuncAttributeMaxDynamicSharedMemorySize, shm_size);
    std::chrono::duration<double> cutlass_totalTime = std::chrono::duration<double>::zero();
    for (int i = 0; i < 100; ++i)
    {
        auto start = std::chrono::high_resolution_clock::now();
        gemm_multi_stage<decltype(gemm_config)>
            <<<grid, block, shm_size>>>(Dptr, Aptr, Bptr, M, N, K);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();
        cutlass_totalTime += end - start;
    }

    hipMemcpy(Dptr_host, Dptr, sizeof(T) * M * N, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    std::cout << "average time: " << cutlass_totalTime.count() * 10 << " ms" << std::endl;
    auto err = hipGetLastError();
    // printf("block = (%d, %d), gird = (%d, %d), shm = %d\n", block.x, block.y,
    //        grid.x, grid.y, shm_size);

    if (err != hipSuccess)
    {
        printf("err = %d, str = %s\n", err, hipGetErrorString(err));
    }

    // auto tD_host = make_tensor(Dptr_host, make_shape(M, N), make_stride(N, 1));
    // auto tile = make_tile(min(8, M), min(8, N));
    // auto t32x32 = local_tile(tD_host, tile, make_coord(0, 0));
    // printf("M = %d, N = %d, K = %d\n", M, N, K);

    // printf("our-impl:\n");
    // print_tensor(t32x32);

    // cublas
    std::cout << "###########################" << std::endl
              << "Run Cublas Gemm:" << std::endl;
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    int cublas_version;
    cublasGetVersion(handle, &cublas_version);
    printf("cublas version = %d\n", cublas_version);
    T *Dptr_cublas;
    T *Dptr_host_blas;
    Dptr_host_blas = (T *)malloc(sizeof(T) * M * N);
    hipMalloc(&Dptr_cublas, sizeof(T) * M * N);
    hipMemset(Dptr_cublas, 0, sizeof(T) * M * N);
    half alpha = 1.f;
    half beta = 0.f;
    std::chrono::duration<double> cublas_totalTime = std::chrono::duration<double>::zero();
    for (int i = 0; i < 100; ++i)
    {
        auto start_cublas = std::chrono::high_resolution_clock::now();
        hipblasStatus_t ret = hipblasHgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, N, M, K,
                                         &alpha, (half *)Bptr, K, (half *)Aptr, K,
                                         &beta, (half *)Dptr_cublas, N);
        hipDeviceSynchronize();
        auto end_cublas = std::chrono::high_resolution_clock::now();
        cublas_totalTime += end_cublas - start_cublas;
        if (ret != HIPBLAS_STATUS_SUCCESS)
        {
            printf("cublas err = %d, str = %s\n", ret, cublasGetStatusString(ret));
        }
    }

    hipMemcpy(Dptr_host_blas, Dptr_cublas, sizeof(T) * M * N,
               hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    std::cout << "cublas average time: " << cublas_totalTime.count() * 10 << " ms" << std::endl;

    // compare
    gpu_compare(Dptr, Dptr_cublas, M * N);
}